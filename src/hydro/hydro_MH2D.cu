#include "hip/hip_runtime.h"
#include "hydro.hpp"

#include <math.h>
#include <iostream>

__device__ double getLimiter(double di1, double di2, double omega);
__device__ double getSlopeX(double* U, int i, int j, int niGhosts, double omega);
__device__ double getSlopeY(double* U, int i, int j, int niGhosts, double omega);

__device__ double calcFluxRho(double rho, double u);
__device__ double calcFluxMom(double rho, double u, double v, double p);
__device__ double calcFluxE(double u, double E, double p);

#define GET(D, I, J) D[(J)*niGhosts + I]

__device__
void Hydro::MUSCLHancock2D(
    double* rhoOld, double* EOld, double* momUOld, double* momVOld,
    int iIndex, int jIndex, int kIndex, int niGhosts,
    double gamma, double dt, double dx, double dy,
    double* rhoNew, double* ENew, double* momUNew, double* momVNew
) {

    double omega = 0.0;

    // 5 Point stencil
    int stencil[5][2] =
    {
        {-1,0}, {0,0}, {1,0},
        {0,-1}, {0,1}
    };


    // Data reconstruction in X
    double rhoL[5];
    double rhoR[5];
    double momUL[5];
    double momUR[5];
    double momVL[5];
    double momVR[5];
    double EL[5];
    double ER[5];
    double rhoD[5];
    double rhoU[5];
    double momUD[5];
    double momUU[5];
    double momVD[5];
    double momVU[5];
    double ED[5];
    double EU[5];

    for (int n=0; n<5; n++) {
        int i = iIndex + stencil[n][0];
        int j = jIndex + stencil[n][1];

        double di = 0.5*getSlopeX(rhoOld, i, j, niGhosts, omega);
        rhoL[n] = GET(rhoOld, i, j) - di;
        rhoR[n] = GET(rhoOld, i, j) + di;

        di = 0.5*getSlopeX(momUOld, i, j, niGhosts, omega);
        momUL[n] = GET(momUOld, i, j) - di;
        momUR[n] = GET(momUOld, i, j) + di;

        di = 0.5*getSlopeX(momVOld, i, j, niGhosts, omega);
        momVL[n] = GET(momVOld, i, j) - di;
        momVR[n] = GET(momVOld, i, j) + di;

        di = 0.5*getSlopeX(EOld, i, j, niGhosts, omega);
        EL[n] = GET(EOld, i, j) - di;
        ER[n] = GET(EOld, i, j) + di;

        di = 0.5*getSlopeY(rhoOld, i, j, niGhosts, omega);
        rhoD[n] = GET(rhoOld, i, j) - di;
        rhoU[n] = GET(rhoOld, i, j) + di;

        di = 0.5*getSlopeY(momUOld, i, j, niGhosts, omega);
        momUD[n] = GET(momUOld, i, j) - di;
        momUU[n] = GET(momUOld, i, j) + di;

        di = 0.5*getSlopeY(momVOld, i, j, niGhosts, omega);
        momVD[n] = GET(momVOld, i, j) - di;
        momVU[n] = GET(momVOld, i, j) + di;

        di = 0.5*getSlopeY(EOld, i, j, niGhosts, omega);
        ED[n] = GET(EOld, i, j) - di;
        EU[n] = GET(EOld, i, j) + di;
    }

    double fx = 0.5*dt/dx;
    double fy = 0.5*dt/dy;
    for (int i=0; i<5; i++) {
        double uL = momUL[i]/rhoL[i];
        double uR = momUR[i]/rhoR[i];

        double vL = momVL[i]/rhoL[i];
        double vR = momVR[i]/rhoR[i];

        double pL = (gamma - 1.0)*(
            EL[i] - 0.5*rhoL[i]*uL*uL
                  - 0.5*rhoL[i]*vL*vL
        );

        double pR = (gamma - 1.0)*(
            ER[i] - 0.5*rhoR[i]*uR*uR
                  - 0.5*rhoR[i]*vR*vR
        );


        double dFx_rho = fx*(calcFluxRho(rhoL[i], uL) -
                             calcFluxRho(rhoR[i], uR));

        double dFx_momN = fx*(calcFluxMom(rhoL[i], uL, uL, pL) -
                              calcFluxMom(rhoR[i], uR, uR, pR));

        double dFx_momT = fx*(calcFluxMom(rhoL[i], uL, vL, 0.0) -
                              calcFluxMom(rhoR[i], uR, vR, 0.0));

        double dFx_E = fx*(calcFluxE(uL, EL[i], pL) -
                           calcFluxE(uR, ER[i], pR));


        double uD = momUD[i]/rhoD[i];
        double uU = momUU[i]/rhoU[i];

        double vD = momVD[i]/rhoD[i];
        double vU = momVU[i]/rhoU[i];

        double pD = (gamma - 1.0)*(
            ED[i] - 0.5*rhoD[i]*uD*uD
                  - 0.5*rhoD[i]*vD*vD
        );

        double pU = (gamma - 1.0)*(
            EU[i] - 0.5*rhoU[i]*uU*uU
                  - 0.5*rhoU[i]*vU*vU
        );


        double dFy_rho = fy*(calcFluxRho(rhoD[i], vD) -
                             calcFluxRho(rhoU[i], vU));

        double dFy_momN = fy*(calcFluxMom(rhoD[i], vD, vD, pD) -
                              calcFluxMom(rhoU[i], vU, vU, pU));

        double dFy_momT = fy*(calcFluxMom(rhoD[i], uD, vD, 0.0) -
                              calcFluxMom(rhoU[i], uU, vU, 0.0));

        double dFy_E = fy*(calcFluxE(vD, ED[i], pD) -
                           calcFluxE(vU, EU[i], pU));


        rhoL[i] += dFx_rho + dFy_rho;
        rhoR[i] += dFx_rho + dFy_rho;
        momUL[i] += dFx_momN + dFy_momT;
        momUR[i] += dFx_momN + dFy_momT;
        momVL[i] += dFx_momT + dFy_momN;
        momVR[i] += dFx_momT + dFy_momN;
        EL[i] += dFx_E + dFy_E;
        ER[i] += dFx_E + dFy_E;

        rhoD[i] += dFx_rho + dFy_rho;
        rhoU[i] += dFx_rho + dFy_rho;
        momUD[i] += dFx_momN + dFy_momT;
        momUU[i] += dFx_momN + dFy_momT;
        momVD[i] += dFx_momT + dFy_momN;
        momVU[i] += dFx_momT + dFy_momN;
        ED[i] += dFx_E + dFy_E;
        EU[i] += dFx_E + dFy_E;

    }


    Flux fluxL, fluxR, fluxU, fluxD;
    double rhoX1 = rhoR[0];
    double uX1 = momUR[0]/rhoR[0];
    double vX1 = momVR[0]/rhoR[0];
    double pX1 = (gamma - 1.0)*(
        ER[0] - 0.5*rhoR[0]*uX1*uX1
              - 0.5*rhoR[0]*vX1*vX1
    );

    double rhoX2 = rhoL[1];
    double uX2 = momUL[1]/rhoL[1];
    double vX2 = momVL[1]/rhoL[1];
    double pX2 = (gamma - 1.0)*(
        EL[1] - 0.5*rhoL[1]*uX2*uX2
              - 0.5*rhoL[1]*vX2*vX2
    );

    double rhoX3 = rhoR[1];
    double uX3 = momUR[1]/rhoR[1];
    double vX3 = momVR[1]/rhoR[1];
    double pX3 = (gamma - 1.0)*(
        ER[1] - 0.5*rhoR[1]*uX3*uX3
              - 0.5*rhoR[1]*vX3*vX3
    );

    double rhoX4 = rhoL[2];
    double uX4 = momUL[2]/rhoL[2];
    double vX4 = momVL[2]/rhoL[2];
    double pX4 = (gamma - 1.0)*(
        EL[2] - 0.5*rhoL[2]*uX4*uX4
              - 0.5*rhoL[2]*vX4*vX4
    );

    Hydro::getFluxHLLC(
        uX1, vX1, rhoX1, pX1,
        uX2, vX2, rhoX2, pX2,
        gamma, &fluxL);

    Hydro::getFluxHLLC(
        uX3, vX3, rhoX3, pX3,
        uX4, vX4, rhoX4, pX4,
        gamma, &fluxR);


    // Y
    double rhoY1 = rhoU[3];
    double uY1 = momUU[3]/rhoU[3];
    double vY1 = momVU[3]/rhoU[3];
    double pY1 = (gamma - 1.0)*(
        EU[3] - 0.5*rhoU[3]*uY1*uY1
              - 0.5*rhoU[3]*vY1*vY1
    );

    double rhoY2 = rhoD[1];
    double uY2 = momUD[1]/rhoD[1];
    double vY2 = momVD[1]/rhoD[1];
    double pY2 = (gamma - 1.0)*(
        ED[1] - 0.5*rhoD[1]*uY2*uY2
              - 0.5*rhoD[1]*vY2*vY2
    );

    double rhoY3 = rhoU[1];
    double uY3 = momUU[1]/rhoU[1];
    double vY3 = momVU[1]/rhoU[1];
    double pY3 = (gamma - 1.0)*(
        EU[1] - 0.5*rhoU[1]*uY3*uY3
              - 0.5*rhoU[1]*vY3*vY3
    );

    double rhoY4 = rhoD[4];
    double uY4 = momUD[4]/rhoD[4];
    double vY4 = momVD[4]/rhoD[4];
    double pY4 = (gamma - 1.0)*(
        ED[4] - 0.5*rhoD[4]*uY4*uY4
              - 0.5*rhoD[4]*vY4*vY4
    );

    Hydro::getFluxHLLC(
        vY1, uY1, rhoY1, pY1,
        vY2, uY2, rhoY2, pY2,
        gamma, &fluxD);

    Hydro::getFluxHLLC(
        vY3, uY3, rhoY3, pY3,
        vY4, uY4, rhoY4, pY4,
        gamma, &fluxU);

    fx = dt/dx;
    fy = dt/dy;

    GET(rhoNew, iIndex, jIndex) = GET(rhoOld, iIndex, jIndex) + fx*(fluxL.rho - fluxR.rho) + fy*(fluxD.rho - fluxU.rho);
    GET(momUNew, iIndex, jIndex) = GET(momUOld, iIndex, jIndex) + fx*(fluxL.momU - fluxR.momU) + fy*(fluxD.momV - fluxU.momV);
    GET(momVNew, iIndex, jIndex) = GET(momVOld, iIndex, jIndex) + fx*(fluxL.momV - fluxR.momV) + fy*(fluxD.momU - fluxU.momU);
    GET(ENew, iIndex, jIndex) = GET(EOld, iIndex, jIndex) + fx*(fluxL.E - fluxR.E) + fy*(fluxD.E - fluxU.E);

}

__device__ double getSlopeX(double* U, int i, int j, int niGhosts, double omega) {
    double di1 = GET(U, i, j) - GET(U, i-1, j);
    double di2 = GET(U, i+1, j) - GET(U, i, j);

    double diU = 0.5*(1.0 + omega)*di1 + 0.5*(1.0 - omega)*di2;

    diU *= getLimiter(di1, di2, omega);

    return diU;
}

__device__ double getSlopeY(double* U, int i, int j, int niGhosts, double omega) {
    double di1 = GET(U, i, j) - GET(U, i, j-1);
    double di2 = GET(U, i, j+1) - GET(U, i, j);

    double diU = 0.5*(1.0 + omega)*di1 + 0.5*(1.0 - omega)*di2;

    diU *= getLimiter(di1, di2, omega);

    return diU;
}

__device__ double getLimiter(double di1, double di2, double omega) {
    double xi;
    // Slope limiter - Van Leer
    if (di2 == 0) {
        xi = 0.0;
    } else {
        double r = di1/di2;
        if (r <= 0.0) {
            xi = 0.0;
        } else {
            double xiR = 2.0/(1.0 - omega + (1 + omega)*r);
            xi = fmin(2*r/(1+r), xiR);
        }
    }
    return xi;
}

__device__
inline double calcFluxRho(double rho, double u) {
    return rho*u;
}


__device__
inline double calcFluxMom(double rho, double u, double v, double p) {
    return rho*u*v + p;
}

__device__
inline double calcFluxE(double u, double E, double p) {
    return u*(E + p);
}
