#include "hip/hip_runtime.h"
#include "mesh/mesh2d.hpp"
#include "hydro/hydro.hpp"

#include <iostream>
#include <omp.h>

#include <fenv.h>


__global__
void copy(double *rhoOut, double *momUOut, double *momVOut, double *EOut,
          double *rhoIn, double *momUIn, double *momVIn, double *EIn,
          int nCells)
{
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int n=index; n<nCells; n += stride) {
        rhoOut[n] = rhoIn[n];
        EOut[n] = EIn[n];
        momUOut[n] = momUIn[n];
        momVOut[n] = momVIn[n];
    }
}


__global__
void evolve(double *rhoIn, double *momUIn, double *momVIn, double *EIn,
          double *rhoOut, double *momUOut, double *momVOut, double *EOut,
          double dt, double dx, double dy, double gamma,
          int nCells, int niGhosts, int ni)
{
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int n=index; n<nCells; n += stride) {
            int j = n/ni;
            int i = n - (ni*j);

            Hydro::MUSCLHancock2D(
                rhoIn, EIn, momUIn, momVIn,
                i+2, j+2, 0, niGhosts,
                gamma, dt, dx, dy,
                rhoOut, EOut, momUOut, momVOut
            );
        }
}


int main(int argc, char* argv[]) {
//     feenableexcept(FE_INVALID | FE_OVERFLOW);

    // Mesh sizes - hardcoded for now
    const int ni = 600;
    const int nj = 240;
    const int problem = 4;

    const double dtOut = 2.0;
    const double tEnd = 5.0;


    const int nCells = ni*nj;

    // MPI environment
    int nprocs, myrank, error;
    nprocs = 1;
    myrank = 0;

    // 100x100 mesh using spherical Sod set-up
    Mesh2D mesh(ni, nj, problem);

    int nCellsGhosts = mesh.niGhosts*mesh.njGhosts;

    if (myrank == 0) {
        mesh.dumpToSILO(0.0, 0);
    }

    double t = 0.0;
    double outNext = t + dtOut;
    int step = 0;

    double *rhoNew;
    double *momUNew;
    double *momVNew;
    double *ENew;

    int allocSize = nCellsGhosts*sizeof(double);
    hipMallocManaged(&rhoNew, allocSize);
    hipMallocManaged(&momUNew, allocSize);
    hipMallocManaged(&momVNew, allocSize);
    hipMallocManaged(&ENew, allocSize);

    for (int i=0; i<mesh.njGhosts*mesh.niGhosts; i++) {
        rhoNew[i] = 0.0001;
        momUNew[i] = 0.0001;
        momVNew[i] = 0.0001;
        ENew[i] = 0.0001;
    }

    mesh.setBoundaries();

    int blockSize = 256;
    int numBlocks = (nCells + blockSize - 1) / blockSize;

    numBlocks = 1;
    blockSize = 256;

    for( ; ; ) {
        step++;

        double dt = 0.013;

        dt = std::min(dt, outNext - t);

        if (myrank == 0) {
            std::cout << "Step: " << step;
            std::cout << ", time = " << t;
            std::cout << ", dt = " << dt << std::endl;
        }


        // Evolve the solution to next time step
        evolve<<<numBlocks, blockSize>>>(
            mesh.rho, mesh.momU, mesh.momV, mesh.E,
            rhoNew, momUNew, momVNew, ENew,
            dt, mesh.dx, mesh.dy, mesh.gamma,
            nCells, mesh.niGhosts, ni
        );

        // Copy into main data arrays
        copy<<<numBlocks, blockSize>>>(
            mesh.rho, mesh.momU, mesh.momV, mesh.E,
            rhoNew, momUNew, momVNew, ENew,
            mesh.niGhosts*mesh.njGhosts
        );

        hipDeviceSynchronize();
//         mesh.setBoundaries();

        t += dt;
        if (t >= outNext) {
            outNext += dtOut;
            if (myrank == 0) mesh.dumpToSILO(t, step);
        }
        if (t > tEnd || step > 100000) break;
    }

    mesh.Kill();

    return 0;
}
